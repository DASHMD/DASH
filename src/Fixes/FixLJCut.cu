#include "hip/hip_runtime.h"
#include "FixLJCut.h"

#include "BoundsGPU.h"
#include "GridGPU.h"
#include "list_macro.h"
#include "State.h"
#include "cutils_func.h"
#include "ReadConfig.h"
#include "EvaluatorWrapper.h"
#include "PairEvaluatorLJ.h"
#include "EvaluatorWrapper.h"
//#include "ChargeEvaluatorEwald.h"
using namespace std;
namespace py = boost::python;
const string LJCutType = "LJCut";



FixLJCut::FixLJCut(boost::shared_ptr<State> state_, string handle_, string mixingRules_)
    : FixPair(state_, handle_, "all", LJCutType, true, false, 1, mixingRules_),
    epsHandle("eps"), sigHandle("sig"), rCutHandle("rCut")
{

    initializeParameters(epsHandle, epsilons);
    initializeParameters(sigHandle, sigmas);
    initializeParameters(rCutHandle, rCuts);
    paramOrder = {rCutHandle, epsHandle, sigHandle};
    readFromRestart();
    canAcceptChargePairCalc = true;
    setEvalWrapper();
}

void FixLJCut::compute(int virialMode) {
    int nAtoms       = state->atoms.size();
    int nPerRingPoly = state->nPerRingPoly;
    int numTypes = state->atomParams.numTypes;
    GPUData &gpd = state->gpd;
    GridGPU &grid = state->gridGPU;
    int activeIdx = gpd.activeIdx();
    uint16_t *neighborCounts = grid.perAtomArray.d_data.data();
    float *neighborCoefs = state->specialNeighborCoefs;
    evalWrap->compute(nAtoms, nPerRingPoly, gpd.xs(activeIdx), gpd.fs(activeIdx),
                      neighborCounts, grid.neighborlist.data(), grid.perBlockArray.d_data.data(),
                      state->devManager.prop.warpSize, paramsCoalesced.data(), numTypes, state->boundsGPU,
                      neighborCoefs[0], neighborCoefs[1], neighborCoefs[2], gpd.virials.d_data.data(), gpd.qs(activeIdx), chargeRCut, virialMode, nThreadPerBlock(), nThreadPerAtom());

}

void FixLJCut::singlePointEng(float *perParticleEng) {
    int nAtoms = state->atoms.size();
    int nPerRingPoly = state->nPerRingPoly;
    int numTypes = state->atomParams.numTypes;
    GPUData &gpd = state->gpd;
    GridGPU &grid = state->gridGPU;
    int activeIdx = gpd.activeIdx();
    uint16_t *neighborCounts = grid.perAtomArray.d_data.data();
    float *neighborCoefs = state->specialNeighborCoefs;
    evalWrap->energy(nAtoms, nPerRingPoly, gpd.xs(activeIdx), perParticleEng, neighborCounts, grid.neighborlist.data(), grid.perBlockArray.d_data.data(), state->devManager.prop.warpSize, paramsCoalesced.data(), numTypes, state->boundsGPU, neighborCoefs[0], neighborCoefs[1], neighborCoefs[2], gpd.qs(activeIdx), chargeRCut, nThreadPerBlock(), nThreadPerAtom());
}

void FixLJCut::singlePointEngGroupGroup(float *perParticleEng, uint32_t tagA, uint32_t tagB) {
    int nAtoms = state->atoms.size();
    int nPerRingPoly = state->nPerRingPoly;
    int numTypes = state->atomParams.numTypes;
    GPUData &gpd = state->gpd;
    GridGPU &grid = state->gridGPU;
    int activeIdx = gpd.activeIdx();
    uint16_t *neighborCounts = grid.perAtomArray.d_data.data();
    float *neighborCoefs = state->specialNeighborCoefs;
    evalWrap->energyGroupGroup(nAtoms, nPerRingPoly, gpd.xs(activeIdx), gpd.fs(activeIdx), perParticleEng, neighborCounts, grid.neighborlist.data(), grid.perBlockArray.d_data.data(), state->devManager.prop.warpSize, paramsCoalesced.data(), numTypes, state->boundsGPU, neighborCoefs[0], neighborCoefs[1], neighborCoefs[2], gpd.qs(activeIdx), chargeRCut, tagA, tagB, nThreadPerBlock(), nThreadPerAtom());
}

void FixLJCut::setEvalWrapper() {
    if (evalWrapperMode == "offload") {
        EvaluatorLJ eval;
        evalWrap = pickEvaluator<EvaluatorLJ, 3, true>(eval, chargeCalcFix);
    } else if (evalWrapperMode == "self") {
        EvaluatorLJ eval;
        evalWrap = pickEvaluator<EvaluatorLJ, 3, true>(eval, nullptr);
    }
}

bool FixLJCut::prepareForRun() {
    //loop through all params and fill with appropriate lambda function, then send all to device
    auto fillGeo = [] (float a, float b) {
        return sqrt(a*b);
    };

    auto fillArith = [] (float a, float b) {
        return (a+b) / 2.0;
    };
    auto fillRCut = [this] (float a, float b) {
        return (float) std::fmax(a, b);
    };
    auto none = [] (float a){};

    auto fillRCutDiag = [this] () {
        return (float) state->rCut;
    };

    auto processEps = [] (float a) {
        return 24*a;
    };
    auto processSig = [] (float a) {
        return pow(a, 6);
    };
    auto processRCut = [] (float a) {
        return a*a;
    };
    prepareParameters(epsHandle, fillGeo, processEps, false);
	if (mixingRules==ARITHMETICTYPE) {
		prepareParameters(sigHandle, fillArith, processSig, false);
	} else {
		prepareParameters(sigHandle, fillGeo, processSig, false);
	}
    prepareParameters(rCutHandle, fillRCut, processRCut, true, fillRCutDiag);

    sendAllToDevice();
    setEvalWrapper();
    return true;
}

string FixLJCut::restartChunk(string format) {
    stringstream ss;
    ss << restartChunkPairParams(format);
    return ss.str();
}


bool FixLJCut::postRun() {

    return true;
}

void FixLJCut::addSpecies(string handle) {
    initializeParameters(epsHandle, epsilons);
    initializeParameters(sigHandle, sigmas);
    initializeParameters(rCutHandle, rCuts);

}

vector<float> FixLJCut::getRCuts() { 
    vector<float> res;
    vector<float> &src = *(paramMap[rCutHandle]);
    for (float x : src) {
        if (x == DEFAULT_FILL) {
            res.push_back(-1);
        } else {
            res.push_back(x);
        }
    }

    return res;
}

void export_FixLJCut() {
    py::class_<FixLJCut, boost::shared_ptr<FixLJCut>, py::bases<FixPair>, boost::noncopyable > (
        "FixLJCut",
        py::init<boost::shared_ptr<State>, string, py::optional<string> > (py::args("state", "handle", "mixingRules"))
    )
      ;

}
