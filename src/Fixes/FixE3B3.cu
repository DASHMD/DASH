#include "hip/hip_runtime.h"
#include "FixE3B3.h"

#include "BoundsGPU.h"
#include "GridGPU.h"
#include "State.h"
#include "boost_for_export.h"
#include "cutils_math.h"
#include "list_macro.h"
#include "EvaluatorE3B3.h"
#include "ThreeBodyE3B3.h"

const std::string E3B3Type = "E3B3";
namespace py = boost::python;
/* Constructor
 * Makes an instance of the E3B3 fix
 */

FixE3B3::FixE3B3(boost::shared_ptr<State> state_,
                  std::string handle_,
                  std::string groupHandle_): Fix(state_, handle_, groupHandle_, E3B3Type, true, true, false, 1) { 
    // set the cutoffs used in this potential
    rf = 5.2; // far cutoff for threebody interactions (Angstroms)
    rs = 5.0; // short cutoff for threebody interactions (Angstroms)
    rc = 7.2; // cutoff for our local neighborlist (Angstroms)
    padding = 2.0; // implied since rc - rf = 2.0; pass this to local GridGPU on instantiation
    // to do: set up the local gridGPU for this set of GPUData; 
    // ---- which means we need to set up the local GPUData;
    // ------- can't do this until we have all the atoms in simulation; so do it in prepareForRun
};

//
// what arguments do we need here? we are updating the molecule positions from 
// the current atom positions

// from FixRigid.cu
__device__ inline float3 positionsToCOM_E3B3(float3 *pos, float *mass, float ims) {
  return (pos[0]*mass[0] + pos[1]*mass[1] + pos[2]*mass[2] + pos[3]*mass[3])*ims;
}

// useful for debugging
__global__ void printGPD_E3B3(uint* ids, float4 *xs, int nMolecules) {
    int idx = GETIDX();
    if (idx < nMolecules) {
        uint id = ids[idx];
        float4 pos = xs[idx];
        printf("molecule id %d at coords %f %f %f\n", id, pos.x, pos.y, pos.z);
    }
}

// prints the global gpd data
__global__ void printGPD_Global(uint* ids, float4 *xs, float4* vs, float4* fs, int nAtoms) {
    int idx = GETIDX();
    if (idx < nAtoms) {
        uint id = ids[idx];
        float4 pos = xs[idx];
        float4 vel = vs[idx];
        float4 force = fs[idx];
        printf("atom id %d at coords %f %f %f with vel %f %f %f and force %f %f %f\n", id, pos.x, pos.y, pos.z, vel.x, vel.y, vel.z, force.x, force.y, force.z);
    }
}

// this function demonstrates how to properly access the neighborlist for E3B3,
// and prints out the results.  It should be found that all values are ~within sqrtf(neighCutSqr)
__global__ void printNlist_E3B3(int* molIdToIdxs,
                                uint* waterMolecIds,
                                int4* atomsFromMolecule,
                                uint16_t* neighborCounts,
                                uint* neighborlist,
                                uint32_t* cumulSumMaxPerBlock,
                                int warpSize,
                                int* idToIdxs,
                                float4* xs,
                                float4* fs, 
                                BoundsGPU bounds,
                                int nMolecules) {
    int idx = GETIDX();
    
    if (idx < nMolecules) {

        //int thisIdx = molIdToIdxs[waterMolecIds[idx]];

        // we need the following function:
        //inline __device__ int baseNeighlistIdxFromIndex(uint32_t *cumulSumMaxPerBlock, int warpSize, int idx) {

        int thisIdx = molIdToIdxs[waterMolecIds[idx]];
        printf("this Idx %d this id %d idx %d", thisIdx, waterMolecIds[idx], idx);
        //int baseIdx = baseNeighlistIdx(cumulSumMaxPerBlock, warpSize);
        int baseIdx = baseNeighlistIdxFromIndex(cumulSumMaxPerBlock, warpSize, thisIdx);
        int numNeighMolecules = neighborCounts[thisIdx];
        //int numNeighMolecules = neighborCounts[idx];

        printf("idx %d baseIdx %d numNeighMolecules %d\n", thisIdx, baseIdx, numNeighMolecules);

        __syncthreads();
        // the neighborCounts and neighborList are for a given idx;
        //int4 atomsMolecule1 = atomsFromMolecule[idx];
        // atomsFromMolecule array is statically sorted by ids
        int4 atomsMolecule1 = atomsFromMolecule[waterMolecIds[idx]];
        int id_O1 = atomsMolecule1.x;
        int idx_a1 = idToIdxs[atomsMolecule1.x];
        int idx_b1 = idToIdxs[atomsMolecule1.y];
        int idx_c1 = idToIdxs[atomsMolecule1.z];

        float4 pos_a1_whole = xs[idx_a1];
        float4 pos_b1_whole = xs[idx_b1];
        float4 pos_c1_whole = xs[idx_c1];

        // now, get just positions in float3
        float3 pos_a1 = make_float3(pos_a1_whole);
        float3 pos_b1 = make_float3(pos_b1_whole);
        float3 pos_c1 = make_float3(pos_c1_whole);

        //int numNeigh = neighborCounts[idx];
        int numNeigh = neighborCounts[thisIdx];

        int counter = 0;
        for (int i = 0; i < numNeigh; i++) {
            int nlistIdx = baseIdx + warpSize*i;
            uint otherIdxRaw = neighborlist[nlistIdx];

            int moleculeIds = waterMolecIds[otherIdxRaw];

            int4 atomsMolecule2 = atomsFromMolecule[moleculeIds];

            int id_O2 = atomsMolecule2.x;
            int id_B2 = atomsMolecule2.y;
            int id_C2 = atomsMolecule2.z;

            int idx_a2 = idToIdxs[atomsMolecule2.x];
            int idx_b2 = idToIdxs[atomsMolecule2.y];
            int idx_c2 = idToIdxs[atomsMolecule2.z];

            float4 pos_a2_whole = xs[idx_a2];
            float4 pos_b2_whole = xs[idx_b2];
            float4 pos_c2_whole = xs[idx_c2];
    
            // here we should extract the positions for the O, H atoms of this water molecule
            float3 pos_a2 = make_float3(pos_a2_whole);
            float3 pos_b2 = make_float3(pos_b2_whole);
            float3 pos_c2 = make_float3(pos_c2_whole);
            
            float3 r_b2a1 = bounds.minImage(pos_b2 - pos_a1);
            float3 r_c2a1 = bounds.minImage(pos_c2 - pos_a1);
            
            float3 r_b1a2 = bounds.minImage(pos_b1 - pos_a2);
            float3 r_c1a2 = bounds.minImage(pos_c1 - pos_a2);

            float r_b2a1_magnitude = length(r_b2a1);
            float r_c2a1_magnitude = length(r_c2a1);
            float r_b1a2_magnitude = length(r_b1a2);
            float r_c1a2_magnitude = length(r_c1a2);

            // we now have our molecule 'j'
            // compute the two-body correction term w.r.t the oxygens
            float3 r_a1a2 = bounds.minImage(pos_a1 - pos_a2);
            float r_a1a2_magnitude = length(r_a1a2);
            counter += 1;
            printf("O atom id %d neighbor ids %d %d %d a1-{a2,b2,c2} distances %f %f %f\n", id_O1, id_O2, id_B2, id_C2, r_a1a2_magnitude, r_b2a1_magnitude, r_c2a1_magnitude);

        }
        printf("finished printing out results for %d of %d neighbors\n", counter,numNeigh);
    }
}


// see FixRigid.cu! does the same thing. but now, we store it in their own gpdLocal..
// -- nothing fancy, dont need the neighborlist here.
__global__ void update_xs(int nMolecules, int4 *waterIds, float4 *mol_xs, int* mol_idToIdxs,
                           float4 *xs, float4 *vs, int *idToIdxs, BoundsGPU bounds) {

     // now do pretty much the same as FixRigid computeCOM()
     // --- remember to account for the M-site, in the event that it has mass
    int idx = GETIDX();
    
    if (idx < nMolecules) {

        // may as well make these arrays

        // just for clarity: we are looking at molecule /id/
        int molId = idx;

        int theseIds[4]; 
        float3 pos[4];
        float mass[4];

        // get the atom /ids/ for molecule id 'idx'
        theseIds[0] = waterIds[molId].x;
        theseIds[1] = waterIds[molId].y;
        theseIds[2] = waterIds[molId].z;
        theseIds[3] = waterIds[molId].w;

        float ims = 0.0f;
        // for each data (pos, vel, force), we need to get the position of atom id at position idToIdx in the global arrays
        for (int i = 0; i < 4; i++) {
            int thisId = theseIds[i];
            int thisIdx = idToIdxs[thisId];
            float3 p = make_float3(xs[thisIdx]);
            pos[i] = p;
            mass[i] = 1.0f / vs[thisIdx].w;
            ims += mass[i];
        }

        ims = 1.0f / ims;
        for (int i = 1; i < 4; i++) {
            float3 delta = pos[i] - pos[0];
            delta = bounds.minImage(delta);
            pos[i] = pos[0] + delta;
        }

        // and here is the COM of our water molecule
        mol_xs[mol_idToIdxs[molId]]  = make_float4(positionsToCOM_E3B3(pos, mass,ims));
        // and corresponding inverse mass
        mol_xs[mol_idToIdxs[molId]].w = ims;

    }

}

void FixE3B3::compute(int VirialMode) {
    
    // send the molecules to the e3b3 evaluator, where we compute both the two-body correction
    // and the three-body interactions.
    // -- send the correct neighbor list (specific to this potential) and the array of water molecules
    //    local to this gpu
    // -- still need to send the global simulation data, which contains the atoms itself
 

    bool computeVirials = false;
    // get the activeIdx for our local gpdLocal (the molecule-by-molecule stuff);
    int activeIdx = gpdLocal.activeIdx();

    // and the global gpd
    // --- IMPORTANT: the virials must be taken from the /global/ gpudata!
    GPUData &gpdGlobal = state->gpd;
    int globalActiveIdx = gpdGlobal.activeIdx();
    
    // our grid data holding our molecule-by-molecule neighbor list
    // -- we need to copy over the molecule array as well.
    
    // although it says 'perAtomArray', note that all of this gpd for this grid is by molecule
    // so, its just a misnomer in this instance. its a count of neighboring molecules.
    

    /* data required for compute_e3b3:
       - nMolecules
       - moleculesIdsToIdxs
       - waterIds (atom IDS in a given molecule)
       - molecules neighborcounts
       - molecules nlist
       - molecules - cumulSumMaxPerBlock (grid.perBlockArray.d_data.data())a
       - warpsize
       - atom idsToIdxs
       - atom positions
       - atom forces (....)
       - boundsGPU (state)
       - virials (global)
       - the evaluator
    */

    if (computeVirials) {
        compute_E3B3<EvaluatorE3B3, true> <<<NBLOCK(nMolecules), PERBLOCK>>> (
            nMolecules, 
            gpdLocal.idToIdxs.d_data.data(), 
            gpdLocal.ids(activeIdx),
            waterIdsGPU.data(),
            gridGPULocal.perAtomArray.d_data.data(),
            gridGPULocal.neighborlist.data(), 
            gridGPULocal.perBlockArray.d_data.data(),
            state->devManager.prop.warpSize,
            gpdGlobal.idToIdxs.d_data.data(), 
            gpdGlobal.xs(globalActiveIdx), 
            gpdGlobal.fs(globalActiveIdx),
            state->boundsGPU, 
            gpdGlobal.virials.d_data.data(),
            evaluator);
    } else {
        compute_E3B3<EvaluatorE3B3, false> <<<NBLOCK(nMolecules), PERBLOCK>>> (
            nMolecules, 
            gpdLocal.idToIdxs.d_data.data(),
            gpdLocal.ids(activeIdx),
            waterIdsGPU.data(),
            gridGPULocal.perAtomArray.d_data.data(),
            gridGPULocal.neighborlist.data(), 
            gridGPULocal.perBlockArray.d_data.data(),
            state->devManager.prop.warpSize,
            gpdGlobal.idToIdxs.d_data.data(), 
            gpdGlobal.xs(globalActiveIdx), 
            gpdGlobal.fs(globalActiveIdx),
            state->boundsGPU, 
            gpdGlobal.virials.d_data.data(),
            evaluator);
    };
}


bool FixE3B3::stepInit(){
    // we use this as an opportunity to re-create the local neighbor list, if necessary
    int periodicInterval = state->periodicInterval;
    
    uint activeIdx = gpdLocal.activeIdx();

    // get the global gpd and the bounds
    uint globalActiveIdx = state->gpd.activeIdx();
    GPUData &gpdGlobal = state->gpd;
    BoundsGPU &bounds = state->boundsGPU;

    if (state->turn % periodicInterval == 0) {
        // do the re-creation of the neighborlist for E3B3
        // -- the xs of the molecules is /not/ updated with the atoms!
        //    but this is what we form our neighborlist off of (for the molecule-by-molecule neighborlist)
        //    so, do a kernel call here to update them to the current positions
        //    of their constituent atoms

        // update the positions of our molecules
        update_xs<<<NBLOCK(nMolecules), PERBLOCK>>>(nMolecules, 
                                                    waterIdsGPU.data(), 
                                                    gpdLocal.xs(activeIdx), 
                                                    gpdLocal.idToIdxs.d_data.data(),
                                                    gpdGlobal.xs(globalActiveIdx), 
                                                    gpdGlobal.vs(globalActiveIdx), 
                                                    gpdGlobal.idToIdxs.d_data.data(),
                                                    bounds
                                                    );
        // for each thread, we have one molecule
        // -- get the atoms for this idx, compute COM, set the xs to the new value, and return
        //    -- need idToIdx for atoms? I think so.  Also, this is easy place to check 
        //       accessing the data arrays

        // pass the local gpdLocal (molecule by molecule) and the global (atom by atom) gpd
        // -- -with this, our local gpdLocal data for the molecule COM is up to date with 
        //     the current atomic data
        hipDeviceSynchronize();
    
        // our grid now operates on the updated molecule xs to get a molecule by molecule neighborlist    
        gridGPULocal.periodicBoundaryConditions();
        
    }

    return true;
}

/* Single Point Eng
 *
 *
 *
 */
//void FixE3B3::singlePointEng(float *perParticleEng) {
    // and, the three body contribution
    // -- we still pass everything molecule by molecule... but add it to their particle arrays

    // gonna need to look up how this is done..
    //return
//}



/* prepareForRun

   */
bool FixE3B3::prepareForRun(){
   
    // as angstroms
    float rs = 5.0;
    float rf = 5.2;

    // E2, Ea, Eb, Ec as kJ/mole -> convert to kcal/mole
    float E2 = 453000;
    float Ea = 150.0000;
    float Eb = -1005.0000;
    float Ec = 1880.0000;

    // k2, k3 as angstroms
    float k2 = 4.872;
    float k3 = 1.907;
    
    float kjToKcal = 0.23900573614;
    E2 *= kjToKcal;
    Ea *= kjToKcal;
    Eb *= kjToKcal;
    Ec *= kjToKcal;

    // 0 = REAL, 1 = LJ (see /src/Units.h)
    if (state->units.unitType == 1) {
        mdError("Units for E3B3 potential are not yet as LJ\n");
    }
        // converting to LJ from kcal/mol

    // instantiate the evaluator
    evaluator = EvaluatorE3B3(rs, rf, E2,
                              Ea, Eb, Ec,
                              k2, k3);
    
    nMolecules = waterMolecules.size();
    waterIdsGPU = GPUArrayDeviceGlobal<int4>(nMolecules);
    waterIdsGPU.set(waterIds.data()); // waterIds vector populated as molecs added
    
    std::vector<float4> xs_vec;
    std::vector<uint> ids;

    xs_vec.reserve(nMolecules);
    ids.reserve(nMolecules);

    
    int workingId = 0;
    for (auto &molecule: waterMolecules)  {
        molecule.id = workingId;
        Vector this_xs = molecule.COM();
        float4 new_xs = make_float4(this_xs[0], this_xs[1], this_xs[2], 0);
        xs_vec.push_back(new_xs);

        ids.push_back(molecule.id);
        workingId++;
    }

    // note that gpd is the /local/ gpd
    gpdLocal.xs.set(xs_vec);
    gpdLocal.ids.set(ids);
   
    std::vector<int> id_vec = LISTMAPREF(Molecule, int, m, waterMolecules, m.id);
    std::vector<int> idToIdxs_vec;
    int size = *std::max_element(id_vec.begin(), id_vec.end()) + 1;
    idToIdxs_vec.reserve(size);
    for (int i=0; i<size; i++) {
        idToIdxs_vec.push_back(-1);
    }
    for (int i=0; i<id_vec.size(); i++) {
        idToIdxs_vec[id_vec[i]] = i;
    }

    gpdLocal.idToIdxsOnCopy = idToIdxs_vec;
    gpdLocal.idToIdxs.set(idToIdxs_vec);
    gpdLocal.xs.dataToDevice();
    gpdLocal.ids.dataToDevice();
    gpdLocal.idToIdxs.dataToDevice();
    int activeIdx = gpdLocal.activeIdx();
    
    double maxRCut = rf;// cutoff of our potential (5.2 A)
    double padding = 1.0;
    double gridDim = maxRCut + padding;

    // this number has no meaning whatsoever; it is completely arbitrary;
    // -- we are not using exclusionMode for this grid or set of GPUData
    int exclusionMode = 30;
    // I think this is doubly irrelevant, since we use a doExclusions(false) method later (below)

    gridGPULocal = GridGPU(state, gridDim, gridDim, gridDim, gridDim, exclusionMode, padding, &gpdLocal);

    // tell gridGPU that the only GPUData we need to sort are positions (and, of course, the molecule/atom id's)
    gridGPULocal.onlyPositions(true);

    // tell gridGPU not to do any exclusions stuff
    gridGPULocal.doExclusions(false);

    // so, the only buffers that we need are the xs and ids!
    gpdLocal.xsBuffer = GPUArrayGlobal<float4>(nMolecules);
    gpdLocal.idsBuffer = GPUArrayGlobal<uint>(nMolecules);
    
    gridGPULocal.periodicBoundaryConditions(-1, true);

    prepared = true;
    return prepared;
}


/* restart chunk?


   */



/* postRun
   * nothing to do here

   */


// the atom ids are presented as the input; assembled into a molecule
void FixE3B3::addMolecule(int id_O, int id_H1, int id_H2, int id_M) {
    
    // id's are arranged as O, H, H, M
    std::vector<int> localWaterIds;

    // add to waterIds vector the four atom ids
    localWaterIds.push_back(id_O);
    localWaterIds.push_back(id_H1);
    localWaterIds.push_back(id_H2);
    localWaterIds.push_back(id_M);

    // mass of O > mass H1 == mass H2 > mass M
    bool ordered = true;
    double massO = state->idToAtom(id_O).mass; 
    double massH1 = state->idToAtom(id_H1).mass;
    double massH2 = state->idToAtom(id_H2).mass;
    double massM = state->idToAtom(id_M).mass;

    // check the ordering
    if (! (massO > massH1 && massO > massH2 )) {
        ordered = false;
    }
    if (massH1 != massH2) ordered = false;
    if (!(massH1 > massM)) ordered = false;

    if (! (ordered)) mdError("Ids in FixE3B3::addMolecule must be as O, H1, H2, M");

    // assemble them in to a molecule
    Molecule thisWater = Molecule(state, localWaterIds);

    // append this molecule to the class variable waterMolecules
    // -- molecule id is implicit as the index in this list
    waterMolecules.push_back(thisWater);

    int4 idsAsInt4 = make_int4(localWaterIds[0], localWaterIds[1], localWaterIds[2], localWaterIds[3]);
    // and add to the global list
    waterIds.push_back(idsAsInt4);


}

/* exports

   */

void export_FixE3B3() {
  py::class_<FixE3B3, boost::shared_ptr<FixE3B3>, py::bases<Fix> > 
	("FixE3B3",
         py::init<boost::shared_ptr<State>, std::string, std::string> 
	 (py::args("state", "handle", "groupHandle")
	 )
	)
    .def("addMolecule", &FixE3B3::addMolecule,
	     (py::arg("id_O"), 
          py::arg("id_H1"), 
          py::arg("id_H2"),
          py::arg("id_M")
         )
    );
}
