#include "hip/hip_runtime.h"

#include "FixDeform.h"
#include "Mod.h"
#include "Interpolator.h"
#include "State.h"
//particles in group handle given will be scaled with the box on deformation
//all and none group handles will scale all or none of the atoms

using std::cout;
using std::endl;

std::string DeformType = "Deform";
FixDeform::FixDeform(boost::shared_ptr<State> state_, std::string handle_,
        std::string groupHandle_, double deformRate_, Vector multiplier_, int applyEvery_) : Fix(state_, handle_, groupHandle_, DeformType, false, false, false, applyEvery_), deformRateInterpolator(deformRate_), multiplier(multiplier_) {
    setPtVolume = -1;
}

FixDeform::FixDeform(boost::shared_ptr<State> state_, std::string handle_,
        std::string groupHandle_, py::object deformRateFunc_, Vector multiplier_, int applyEvery_) : Fix(state_, handle_, groupHandle_, DeformType, false, false, false, applyEvery_), deformRateInterpolator(deformRateFunc_), multiplier(multiplier_)  {
    setPtVolume = -1;
}

FixDeform::FixDeform(boost::shared_ptr<State> state_, std::string handle_,
        std::string groupHandle_, py::list intervals_, py::list rates_, Vector multiplier_, int applyEvery_)  : Fix(state_, handle_, groupHandle_, DeformType, false, false, false, applyEvery_), deformRateInterpolator(intervals_, rates_), multiplier(multiplier_)  {
    setPtVolume = -1;
}


bool FixDeform::prepareForRun() {
    deformRateInterpolator.turnBeginRun = state->runInit;
    deformRateInterpolator.turnFinishRun = state->runInit + state->runningFor;
    if (setPtVolume != -1) {
        //then override entered rate
        double curVol = state->bounds.volume();
        double volRatio = setPtVolume / curVol;
        int nDimDeform = 0;
        for (int i=0; i<3; i++) {
            nDimDeform += multiplier[i]>0 ? 1 : 0;
        }
        double sideLenRatio = pow(volRatio, 1.0/nDimDeform);
        for (int i=0; i<3; i++) {
            if (multiplier[i]>0) {
                multiplier[i] = state->bounds.rectComponents[i] / state->bounds.rectComponents[0];
            }
        }
        double rate = -state->bounds.rectComponents[0] * (1-sideLenRatio) / (state->dt * state->runningFor);
        deformRateInterpolator = Interpolator(rate);
        

    }
    return true;
}

bool FixDeform::stepFinal() {
    deformRateInterpolator.computeCurrentVal(state->turn);
    double rate = deformRateInterpolator.getCurrentVal();
    float3 deltaBounds = (multiplier * rate * state->dt).asFloat3();
    float3 newTrace = state->boundsGPU.rectComponents + deltaBounds;
    float3 scaleBy = newTrace / state->boundsGPU.rectComponents;
    Mod::scaleSystem(state, scaleBy, groupTag);
    return true;

}

void FixDeform::toVolume(double volume) {
    setPtVolume = volume;
}

void export_FixDeform()
{
    py::class_<FixDeform,                    // Class
               boost::shared_ptr<FixDeform>, // HeldType
               py::bases<Fix>,                   // Base class
               boost::noncopyable>
    (
        "FixDeform",
        py::init<boost::shared_ptr<State>, std::string, std::string, py::object, py::optional<Vector, int> >(
            py::args("state", "handle", "groupHandle", "deformFunc", "multiplier", "applyEvery")
        )
    )
    .def(py::init<boost::shared_ptr<State>, std::string, std::string, py::list, py::list, py::optional<Vector, int> >(
                py::args("state", "handle", "groupHandle", "intervals", "deformRates", "multiplier", "applyEvery")

                )
        )
    .def(py::init<boost::shared_ptr<State>, std::string, std::string, double, py::optional<Vector, int> >(
                py::args("state", "handle", "groupHandle", "deformRate", "multiplier", "applyEvery")

                )
        )
    .def("toVolume", &FixDeform::toVolume)

    ;
}

