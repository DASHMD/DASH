#include "hip/hip_runtime.h"
#include "Fix2d.h"
#include "State.h"

void __global__ compute_cu(float4 *xs, float4 *vs, float4 *fs, int nAtoms) {
    int idx = GETIDX();
    if (idx < nAtoms) {
        xs[idx].z = 0;
        vs[idx].z = 0;
        fs[idx].z = 0;

    }
}
//THIS NEEDS TO GO LAST

void Fix2d::compute(int virialMode) {
    //going to zero z in xs, vs, fs
    int nAtoms = state->atoms.size();
    GPUData &gpd = state->gpd;
    compute_cu<<<NBLOCK(nAtoms), PERBLOCK>>>(gpd.xs.getDevData(), gpd.vs.getDevData(), gpd.fs.getDevData(), nAtoms);
}

void export_Fix2d() {
    boost::python::class_<Fix2d,
                          boost::shared_ptr<Fix2d>,
                          boost::python::bases<Fix> >(
        "Fix2d",
        boost::python::init<boost::shared_ptr<State>, std::string, int>(
            boost::python::args("state", "handle", "applyEvery"))
    )
    ;
}
