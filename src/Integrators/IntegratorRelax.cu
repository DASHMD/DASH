#include "hip/hip_runtime.h"
#include "IntegratorRelax.h"
#include "cutils_func.h"
#include "State.h"

#include "Fix.h"
using namespace MD_ENGINE;

IntegratorRelax::IntegratorRelax(SHARED(State) state_)
    : Integrator(state_.get())
{
    //FIRE parameters
    alphaInit = 0.1;
    alphaShrink = 0.99;
    dtGrow = 1.1;
    dtShrink = 0.5;
    delay = 5;
    dtMax_mult = 10;
}

//kernels for FIRE relax
//VDotF by hand
__global__ void vdotF_cu(float *dest, float4 *vs,float4 *fs, int n) {
    extern __shared__ float tmp[];  // should have length of # threads in a block (PERBLOCK)
    int potentialIdx = blockDim.x*blockIdx.x + threadIdx.x;
    if (potentialIdx < n) {
        tmp[threadIdx.x] = dot(make_float3(vs[blockDim.x*blockIdx.x + threadIdx.x]),
                               make_float3(fs[blockDim.x*blockIdx.x + threadIdx.x]));
    } else {
        tmp[threadIdx.x] = 0;
    }
    __syncthreads();
    int maxLookahead = log2f(blockDim.x-1);
    for (int i=0; i<=maxLookahead; i++) {
        int curLookahead = powf(2, i);
        if (! (threadIdx.x % (curLookahead*2))) {
            tmp[threadIdx.x] += tmp[threadIdx.x + curLookahead];
        }
        __syncthreads();
    }
    if (threadIdx.x == 0) {
        atomicAdd(dest, tmp[0]);
    }
}

//update velocities
__global__ void FIRE_new_vel_cu(int nAtoms, float4 *vs, float4 *fs, float scale1, float scale2) {
    int idx = GETIDX();
    if (idx < nAtoms) {
        float4 vel = vs[idx];
        float4 force = fs[idx];
        float invmass = vel.w;
        float4 newVel = vel*scale1 + force*scale2;
        newVel.w = invmass;
        vs[idx] = newVel;
    }
}

//zero velocities
__global__ void zero_vel_cu(int nAtoms, float4 *vs) {
    int idx = GETIDX();
    if (idx < nAtoms) {
        float4 vel = vs[idx];
        vs[idx] = make_float4(0.0f,0.0f,0.0f,vel.w);
    }
}

//MD step
__global__ void FIRE_preForce_cu(int nAtoms, float4 *xs, float4 *vs, float4 *fs, float dt, float dtf) {
    int idx = GETIDX();
    if (idx < nAtoms) {


        float4 vel = vs[idx];
        float4 force = fs[idx];

        float invmass = vel.w;
        float groupTag = force.w;
        xs[idx] = xs[idx] + make_float3(vel) * dt;
        float3 newVel = make_float3(force) * dtf * invmass;
        vs[idx] = vel + newVel;
        fs[idx] = make_float4(0, 0, 0, groupTag);
    }
}




double IntegratorRelax::run(int numTurns, double fTol) {
    std::cout << "FIRE relaxation\n";
    basicPreRunChecks();  
    std::vector<bool> prepared = basicPrepare(numTurns);
    force(true);

    for (int i = 0; i<prepared.size(); i++) {
        if (!prepared[i]) {
            for (Fix *f : state->fixes) {
                bool isPrepared = f->prepareForRun();
                if (!isPrepared) {
                    mdError("A fix is unable to be instantiated correctly.");
                }
            }
        }
    }

    CUT_CHECK_ERROR("FIRE relaxation init failed");  // Debug feature, checks error code

    //initial  values
    int lastNegative = 0;
    double dt = state->dt;
    double alpha = alphaInit;
    const double dtMax = dtMax_mult * dt;

    int warpSize = state->devManager.prop.warpSize;

    //assuming constant number of atoms during run
    int atomssize = state->atoms.size();
    int periodicInterval = state->periodicInterval;
    int nblock = NBLOCK(atomssize);
    int remainder = state->turn % periodicInterval;
    int turnInit = state->turn; 

    //set velocity to 0
    // 	state->gpd.vs.memsetByVal(make_float3(0.0f,0.0f,0.0f);
    zero_vel_cu<<<nblock, PERBLOCK>>>(atomssize,state->gpd.vs.getDevData());
    CUT_CHECK_ERROR("zero_vel_cu kernel execution failed");

    //vars to store kernels outputs
    GPUArrayGlobal<float>VDotV(1);
    GPUArrayGlobal<float>VDotF(1);
    GPUArrayGlobal<float>FDotF(1);
    GPUArrayGlobal<float>force(1);

    //neighborlist build
    state->gridGPU.periodicBoundaryConditions(-1, true);
    DataManager &dataManager = state->dataManager;
    for (int i=0; i<numTurns; i++) {
        checkQuit();
        //init to 0 on cpu and gpu
        VDotV.memsetByVal(0.0);
        VDotF.memsetByVal(0.0);
        FDotF.memsetByVal(0.0);
        int virialMode = dataManager.getVirialModeForTurn(state->turn);
        stepInit(virialMode==1 or virialMode==2);
        //vdotF calc
        if (! ((remainder + i) % periodicInterval)) {
            state->gridGPU.periodicBoundaryConditions();
        }
        asyncOperations();

        vdotF_cu <<<nblock,PERBLOCK,sizeof(float)*PERBLOCK>>>(
                    VDotF.getDevData(),
                    state->gpd.vs.getDevData(),
                    state->gpd.fs.getDevData(),
                    atomssize);
        CUT_CHECK_ERROR("vdotF_cu kernel execution failed");
        VDotF.dataToHost();

        if (VDotF.h_data[0] > 0) {
            //VdotV calc
            accumulate_gpu<float, float4, SumVectorSqr3D, N_DATA_PER_THREAD> <<<NBLOCK(atomssize / (double) N_DATA_PER_THREAD), PERBLOCK, N_DATA_PER_THREAD*sizeof(float)*PERBLOCK>>> 
                (
                 VDotV.getDevData(),
                 state->gpd.vs.getDevData(),
                 atomssize,
                 warpSize,
                 SumVectorSqr3D()
                );


            /*
            sumVectorSqr3D<float,float4, N_DATA_PER_THREAD> <<<NBLOCK(atomssize/(double)N_DATA_PER_THREAD),PERBLOCK,N_DATA_PER_THREAD*sizeof(float)*PERBLOCK>>>(
                                            VDotV.getDevData(),
                                            state->gpd.vs.getDevData(),
                                            atomssize,
                                            warpSize);
                                            */
            CUT_CHECK_ERROR("vdotV_cu kernel execution failed");
            VDotV.dataToHost();

            //FdotF
            accumulate_gpu<float, float4, SumVectorSqr3D, N_DATA_PER_THREAD> <<<NBLOCK(atomssize / (double) N_DATA_PER_THREAD), PERBLOCK, N_DATA_PER_THREAD*sizeof(float)*PERBLOCK>>> 
                (
                 FDotF.getDevData(),
                 state->gpd.fs.getDevData(),
                 atomssize,
                 warpSize,
                 SumVectorSqr3D()
                );

            /*
            sumVectorSqr3D<float,float4, N_DATA_PER_THREAD> <<<NBLOCK(atomssize/(double)N_DATA_PER_THREAD),PERBLOCK,N_DATA_PER_THREAD*sizeof(float)*PERBLOCK>>>(
                                            FDotF.getDevData(),
                                            state->gpd.fs.getDevData(),
                                            atomssize,
                                            warpSize);
                                            */
            CUT_CHECK_ERROR("fdotF_cu kernel execution failed");
            FDotF.dataToHost();

            float scale1 = 1 - alpha;
            float scale2 = 0;
            hipDeviceSynchronize();
            if (FDotF.h_data[0] != 0) {
                scale2 = alpha * sqrt(VDotV.h_data[0] / FDotF.h_data[0]);
            }
            //set velocity to
            //a.vel = a.vel * scale1 + a.force * scale2;
            FIRE_new_vel_cu <<<nblock, PERBLOCK>>>(
                                atomssize,
                                state->gpd.vs.getDevData(),
                                state->gpd.fs.getDevData(),
                                scale1,scale2);
            //check number of steps since negative 
            if (i - lastNegative > delay) {
                dt = fmin(dt*dtGrow, dtMax);
                alpha *= alphaShrink;

            }
        } else {
            lastNegative = i;
            dt *= dtShrink;
            alpha = alphaInit;
            //set velocity to 0
            //state->gpd.vs.memsetByVal(make_float3(0.0f,0.0f,0.0f);
            zero_vel_cu <<<nblock, PERBLOCK>>>(atomssize, state->gpd.vs.getDevData());
            CUT_CHECK_ERROR("zero_vel_cu kernel execution failed");

        }

        FIRE_preForce_cu <<<nblock, PERBLOCK>>>(
                            atomssize,
                            state->gpd.xs.getDevData(),
                            state->gpd.vs.getDevData(),
                            state->gpd.fs.getDevData(),
                            dt, dt*state->units.ftm_to_v);
        CUT_CHECK_ERROR("FIRE_preForce_cu kernel execution failed");

        Integrator::forceSingle(virialMode);

        if (fTol > 0 and i > delay and not (i%delay)) { //only check every so often
            //total force calc
            force.memsetByVal(0.0);
            accumulate_gpu<float, float4, SumVectorSqr3D, N_DATA_PER_THREAD> <<<NBLOCK(atomssize/(double)N_DATA_PER_THREAD),PERBLOCK,N_DATA_PER_THREAD*sizeof(float)*PERBLOCK>>> 
                (
                 force.getDevData(),
                 state->gpd.fs.getDevData(),
                 atomssize,
                 warpSize,
                 SumVectorSqr3D()
                );
            /*
            sumVectorSqr3D<float,float4, N_DATA_PER_THREAD> <<<NBLOCK(atomssize/(double)N_DATA_PER_THREAD),PERBLOCK,N_DATA_PER_THREAD*sizeof(float)*PERBLOCK>>>(
                                        force.getDevData(),
                                        state->gpd.fs.getDevData(),
                                        atomssize,
                                        warpSize);
                                        */
            CUT_CHECK_ERROR("kernel execution failed");//Debug feature, check error code

            force.dataToHost();
            //std::cout<<"Fire relax: force="<<force<<"; turns="<<i<<'\n';
            hipDeviceSynchronize();

            if (force.h_data[0] < fTol*fTol) {//tolerance achived, exting
                basicFinish();
                float finalForce = sqrt(force.h_data[0]);
                std::cout<<"FIRE relax done: force="<< finalForce <<"; turns="<<i+1<<'\n';
                return finalForce;
            }
        } 

        //shout status
        if (state->verbose and not ((state->turn - turnInit) % state->shoutEvery)) {
            std::cout << "Turn " << (int) state->turn 
                      << " " << (int) (100 * (state->turn - turnInit) / (double) numTurns)
                      << " percent done" << std::endl;
        }
        dataManager.clearVirialTurn(state->turn);
        state->turn++;

    }
    //total force calculation
    force.memsetByVal(0.0);
    accumulate_gpu<float, float4, SumVectorSqr3D, N_DATA_PER_THREAD> <<<NBLOCK(atomssize/(double)N_DATA_PER_THREAD),PERBLOCK,N_DATA_PER_THREAD*sizeof(float)*PERBLOCK>>> 
        (
         force.getDevData(),
         state->gpd.fs.getDevData(),
         atomssize,
         warpSize,
         SumVectorSqr3D()
        );
    /*
    sumVectorSqr3D<float,float4, N_DATA_PER_THREAD> <<<NBLOCK(atomssize/(double)N_DATA_PER_THREAD),PERBLOCK,N_DATA_PER_THREAD*sizeof(float)*PERBLOCK>>>(
                                  force.getDevData(),
                                  state->gpd.fs.getDevData(),
                                  atomssize,
                                  warpSize);
                                  */
    CUT_CHECK_ERROR("kernel execution failed"); //Debug feature, check error code

    basicFinish();
    hipDeviceSynchronize();
    float finalForce = sqrt(force.h_data[0]) / atomssize;
    std::cout << "FIRE relax done: force=" << finalForce 
              << "; turns=" << numTurns << std::endl;

    return finalForce;
}

void export_IntegratorRelax() {
    boost::python::class_<IntegratorRelax,
                          SHARED(IntegratorRelax),
                          boost::python::bases<Integrator>,
                          boost::noncopyable >(
            "IntegratorRelax",
            boost::python::init<SHARED(State)>()
    )
    .def("run", &IntegratorRelax::run)
    .def("set_params", &IntegratorRelax::set_params,
            (boost::python::arg("alphaInit")=-1,
             boost::python::arg("alphaShrink")=-1,
             boost::python::arg("dtGrow")=-1,
             boost::python::arg("dtShrink")=-1,
             boost::python::arg("delay")=-1,
             boost::python::arg("dtMax_mult")=-1)
    )
    ;
}

