#include "hip/hip_runtime.h"
#include "DataComputerCOMV.h"
#include "cutils_func.h"
#include "boost_for_export.h"
#include "State.h"
namespace py = boost::python;
using namespace MD_ENGINE;

// scalar, because we just need the one return - not a per-atom thing.
DataComputerCOMV::DataComputerCOMV(State *state_) : DataComputer(state_, "scalar", false) {
    sumMomentum = GPUArrayGlobal<float4>(2);
}


void DataComputerCOMV::computeScalar_GPU(bool transferToCPU, uint32_t groupTag) {
    GPUData &gpd = state->gpd;
    sumMomentum.d_data.memset(0);
    lastGroupTag = groupTag;
    int nAtoms = state->atoms.size();

    accumulate_gpu<float4, float4, SumVectorXYZOverW, N_DATA_PER_THREAD> <<<NBLOCK(nAtoms / (double) N_DATA_PER_THREAD), PERBLOCK, N_DATA_PER_THREAD*PERBLOCK*sizeof(float4)>>>
            (
             sumMomentum.getDevData(),
             gpd.vs.getDevData(),
             nAtoms,
             state->devManager.prop.warpSize,
             SumVectorXYZOverW()
            );
    
    if (transferToCPU) {
        //does NOT sync
        sumMomentum.dataToHost();
    }
}


void DataComputerCOMV::prepareForRun() {
    DataComputer::prepareForRun();
    //then my own stuff
}


void DataComputerCOMV::computeScalar_CPU() {
    systemMomentum = sumMomentum.h_data[0];

}

void DataComputerCOMV::appendScalar(boost::python::list &vals) {
    //Virial tmp = Virial(systemMomentum.x, systemMomentum.y, systemMomentum.z,
    //                    systemMomentum.w, 0, 0);
    // boost is an enigma
    std::vector<double> tmp = std::vector<double>(4,0);
    tmp[0] = systemMomentum.x;
    tmp[1] = systemMomentum.y;
    tmp[2] = systemMomentum.z;
    tmp[3] = systemMomentum.w;
    //vals.append(systemMomentum);
    vals.append(tmp);
}
